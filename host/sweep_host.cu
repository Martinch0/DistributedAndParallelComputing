#include "hip/hip_runtime.h"
/// COMMENT

#include "cutil_inline.h"
#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <cstdlib>
#include <vector>
using namespace std;

float** h_ellipse_vertex;
float* h_torus_vertex;
float* h_torus_surface;
float* sweep_origin;

int number_sweep_steps;
int number_ellipse_points;

// Count the lines in a file
int countLines(char *filename) {
	ifstream fin(filename);
	int input_size = count(istreambuf_iterator<char>(fin), istreambuf_iterator<char>(), '\n');
	fin.seekg(ios::beg);
	return input_size;
}

// Read from file
float** readFromFile(char *filename)
{
	ifstream fin(filename);
	int len = countLines(filename);
	number_ellipse_points = len;
	float **arr = new float*[len];
	for(int i=0; i<len; i++) {
		arr[i] = new float[4];
		fin>>arr[i][0]>>arr[i][1]>>arr[i][2];
		arr[i][3] = 1.0f;
	}
	return arr;
}

// Write to file
void writeToFile(char *filename, float** arr, int x, int y)
{
	ofstream fout(filename);
	for(int i = 0; i<y; i++) {
		for(int j=0; j<x; j++) {
			fout<<arr[i][j]<<' ';
    }
		fout<<endl;
  }
	fout.flush();
	fout.close();
}

// Write to console
void writeToConsole(float** arr, int x, int y) {
	for(int i = 0; i<y; i++) {
		for(int j=0; j<x; j++) {
			cout<<arr[i][j]<<' ';
    }
		cout<<endl;
  }
}

// Matrix multiplication
// X is the number of ROWS, Y is the number of COLS.
float** matrix_mul(float **a, float **b, int ax, int ay, int bx, int by)
{
	float** result = new float * [ax];
  //init array
  for(int i=0; i < ax; i++) {
    result[i] = new float[by];
  }
  //for every row in the result
  for(int i=0; i < ax; i++) {
    //for every column in the result
    for(int j=0; j < by; j++) {
      float sum = 0;
      //find the sum of the multiplied row and column
      for(int k=0; k < ay; k++) {
        sum += a[i][k] * b[k][j];
      }
      result[i][j] = sum;
    }
  }
	return result;
}

// Rotation Transformation Matrix
float** rotation_matrix(int angle)
{
	float** rotation = new float*[4];
	rotation[0] = new float[4];
  rotation[0][0] = cos(angle);
	rotation[0][1] = 0;
  rotation[0][2] = sin(angle);
  rotation[0][3] = 0;
	rotation[1] = new float[4];
  rotation[1][0] = rotation[1][2] = rotation[1][3] = 0;
	rotation[1][1] = 1;
	rotation[2] = new float[4];
  rotation[2][0] = -sin(angle);
	rotation[2][1] = rotation[2][3] = 0;
  rotation[2][2] = cos(angle);
	rotation[3] = new float[4];
  rotation[3][0] = rotation[3][1] = rotation[3][2] = 0;
  rotation[3][3] = 1;
  return rotation;
}

// Sweep the ellipse
void sweep()
{
	
}

// Generate the surface table
void generateSurfaceTable()
{
  //Yu-Yang: assumming rings are one after the other.
}

int main(int argc, char** argv)
{
	//CUDA properties
	int devID;
	hipDeviceProp_t props;

	// get number of SMs on this GPU
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
  
	cout<<"Start reading"<<endl;
	float **h_ellipse_vertex = readFromFile("../ellipse_matrix.txt");
	//for(int i=0; i<number_ellipse_points; i++) {
	//	cout<<h_ellipse_vertex[i][0]<<' '<<h_ellipse_vertex[i][1]<<' '<<h_ellipse_vertex[i][2]<<' '<<h_ellipse_vertex[i][3]<<endl;
	//}


  float ** arr1 = new float*[2];
  arr1[0] = new float[2];
  arr1[1] = new float[2];

  arr1[0][0] = 1;
  arr1[0][1] = 2;
  arr1[1][0] = 3;
  arr1[1][1] = 4;


  float ** arr2 = new float*[2];
  arr2[0] = new float[2];
  arr2[1] = new float[2];

  arr2[0][0] = 4;
  arr2[0][1] = 3;
  arr2[1][0] = 2;
  arr2[1][1] = 1;


  float ** arr3 = matrix_mul(arr1, arr2, 2, 2, 2, 2);


  writeToConsole(rotation_matrix(45), 2, 2);

  //
  // INIT DATA HERE
  //
  
  // print information
  cout << "Number of ellipse vertices : " << number_ellipse_points << endl;
  cout << "Number of rotational sweep steps : " << number_sweep_steps << endl;
  cout << "Rotational sweep origin : " << "[" << sweep_origin[0] << ", " << sweep_origin[1] << ", " << sweep_origin[2] << "]" << endl;
  
  // create a timer
	unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
	// start the timer
  cutilCheckError(cutStartTimer(timer));

	//
	//
	// DO STUFF HERE
	//
	//

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed");

	// wait for device to finish
	hipDeviceSynchronize();

	cutilCheckError(cutStopTimer(timer));

	// exit and clean up device status
	hipDeviceReset();

	return 0;
}
