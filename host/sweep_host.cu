/// COMMENT

#include "cutil_inline.h"
#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <cstdlib>
#include <vector>
using namespace std;

float** h_ellipse_vertex;
float* h_torus_vertex;
float* h_torus_surface;
float* sweep_origin;

int number_sweep_steps;
int number_ellipse_points;

// Count the lines in a file
int countLines(char *filename) {
	ifstream fin(filename);
	int input_size = count(istreambuf_iterator<char>(fin), istreambuf_iterator<char>(), '\n');
	fin.seekg(ios::beg);
	return input_size;
}

// Read from file
float** readFromFile(char *filename)
{
	ifstream fin(filename);
	int len = countLines(filename);
	number_ellipse_points = len;
	float **arr = new float*[len];
	for(int i=0; i<len; i++) {
		arr[i] = new float[4];
		fin>>arr[i][0]>>arr[i][1]>>arr[i][2];
		arr[i][3] = 1.0f;
	}
	return arr;
}

// Write to file
void writeToFile(char *filename, float** arr, int x, int y)
{
	ofstream fout(filename);
	for(int i = 0; i<y; i++) {
		for(int j=0; j<x; j++) {
			fout<<arr[i][j]<<' ';
    }
		fout<<endl;
  }
	fout.flush();
	fout.close();
}

// Write to console
void writeToConsole(float** arr, int x, int y) {
	for(int i = 0; i<y; i++) {
		for(int j=0; j<x; j++) {
			cout<<arr[i][j]<<' ';
    }
		cout<<endl;
  }
}

// Matrix multiplication
float** matrix_mul(float **a, float **b, int ax, int ay, int bx, int by)
{
  cout << "bleh12321321" << endl;
	float** result = new float * [ax];

  for(int i=0; i < ax; i++) {
    result[i] = new float[by];
  }

  for(int i=0; i < ax; i++) {
    for(int j=0; j < by; j++) {
      float sum = 0;
      for(int k=0; k < ax; k++) {
        sum += a[ax][k] * b[k][ax];
      }
      result[i][j] = sum;
    }
  }

	return result;
}

// Sweep the ellypse
void sweep()
{
  //Martin: do one ring after the other.
}

// Generate the surface table
void generateSurfaceTable()
{
  //Yu-Yang: assumming rings are one after the other.
}

int main(int argc, char** argv)
{
	//CUDA properties
	int devID;
	hipDeviceProp_t props;

	// get number of SMs on this GPU
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&props, devID));
  
	cout<<"Start reading"<<endl;
	float **h_ellipse_vertex = readFromFile("../ellipse_matrix.txt");
	//for(int i=0; i<number_ellipse_points; i++) {
	//	cout<<h_ellipse_vertex[i][0]<<' '<<h_ellipse_vertex[i][1]<<' '<<h_ellipse_vertex[i][2]<<' '<<h_ellipse_vertex[i][3]<<endl;
	//}

  cout << "bleh" << endl;

  float ** arr1 = new float * [1];
  arr1[0] = new float[4];
  arr1[0][0] = 1;
  arr1[0][1] = 1;
  arr1[0][2] = 1;
  arr1[0][3] = 1;


  cout << "bleh2" << endl;

  float ** arr2 = new float * [4];
  arr2 [0] = new float[1];
  arr2 [1] = new float[1];
  arr2 [2] = new float[1];
  arr2 [3] = new float[1];

  arr2[0][0] = 1;
  arr2[1][0] = 1;
  arr2[2][0] = 1;
  arr2[3][0] = 1;

  cout << "bleh3" << endl;

  float ** arr3 = matrix_mul(arr1, arr2, 4, 1, 1, 4);

  cout << "bleh4" << endl;

  writeToConsole(arr3, 1, 4);

  //
  // INIT DATA HERE
  //
  
  // print information
  cout << "Number of ellipse vertices : " << number_ellipse_points << endl;
  cout << "Number of rotational sweep steps : " << number_sweep_steps << endl;
  cout << "Rotational sweep origin : " << "[" << sweep_origin[0] << ", " << sweep_origin[1] << ", " << sweep_origin[2] << "]" << endl;
  
  // create a timer
	unsigned int timer = 0;
  cutilCheckError(cutCreateTimer(&timer));
	// start the timer
  cutilCheckError(cutStartTimer(timer));

	//
	//
	// DO STUFF HERE
	//
	//

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed");

	// wait for device to finish
	hipDeviceSynchronize();

	cutilCheckError(cutStopTimer(timer));

	// exit and clean up device status
	hipDeviceReset();

	return 0;
}
